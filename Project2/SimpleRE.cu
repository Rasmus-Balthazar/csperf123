#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

// Input, how much is left of input, Pattern, Pattern length
/** Progression
 * Literal Search (ignorant) https://github.com/cli117/thesis_work/blob/main/literal_match_normal/literal_match.cu
 * Begin adding rules - only extract relevant RE patterns as Literal Search
 * Wildcards
 * Repetitions
 * Ranges/Sets
 * Ors/Options
 */

/**
 * How you doing?
 */

typedef struct {
    int start_index;
    int length;
    int pattern_idx;
} Match;

__device__ int matches(char pattern, char text);

__global__ void simple_gpu_re(char *text, int text_len, int pattern_count, char *patterns[], int patterns_len[], unsigned int matches_found[], Match match_arr[]) {
    if (threadIdx.x == 0) {
        for (int i = 0; i < 3; i++)
        {
            printf("pattern %i: %s", i, patterns[i]);
        }
    }

    int pattern_len = patterns_len[blockIdx.x];
    int stride = blockDim.x;
    for (int pattern_index = blockIdx.x; pattern_index < gridDim.x; pattern_index += gridDim.x) {
        char *pattern = patterns[pattern_index];
        for (int i = threadIdx.x; i < text_len; i += stride) {
            int pattern_off = 0;
            int text_off = 0;
            int does_match;
            do
            {
                does_match = matches(pattern[pattern_off], text[i + text_off]);
                pattern_off+= does_match;
                text_off+= does_match;
                // If the offset is longer than the pattern length we have found it
                if (pattern_off >= pattern_len) {
                    printf("Matched pattern \"%s\" on thread %i\n", pattern, threadIdx.x);
                    unsigned int val = matches_found[pattern_index];
                    // We are relying on the checks not being exhaustive by doing val > i before atomicCAS
                    while (val > i && atomicCAS(matches_found + pattern_index, val, i) > i) {
                        val = matches_found[pattern_index];
                        // Compares b to a, and if true then writes c into a.
                    }
                    break;
                }
            } while (does_match);
            
            if ((i + stride) > matches_found[pattern_index] || (i+stride) > text_len) 
            {
                // If match here, collection process can start,
                __syncthreads(); // Synchronize threads in the block
                if (threadIdx.x == matches_found[pattern_index]%stride) {
                    match_arr[pattern_index].start_index = i;
                    match_arr[pattern_index].length = text_off;
                    match_arr[pattern_index].pattern_idx = pattern_index;

                    printf("Match for pattern \"%s\" found at %i\n", patterns[pattern_index], i);
                }
                break;
            }
        }
    }
}

// Update this to work with tokens, and return how much of text was consumed
__device__ int matches(char pattern, char text) {
    printf("Trying to match %c and %c\n", pattern, text);
    return pattern == text;
}

__global__ void print_char_array(char* array, int array_size) {
    for(int i = 0; i < array_size; i++) {
        printf("Char at index: %i was %c", i, array[i]);
    }
}

#define BLOCK_SIZE 256  // Number of threads per block
#define ARRAY_SIZE 1024  // Size of the input arrays

int main() {
    //h_ for host 
    char* h_text = "dette er en lang test tekst xD";
    int text_len = strlen(h_text);
    char* h_patterns[] = { "test", "er", "nope" };
    int h_pattern_lens[] = {4, 2, 4}; //because of terminating char
    unsigned int h_matches_found[] = {-1u, -1u, -1u};
    Match* h_match_arr = (Match*)calloc(3, sizeof(Match)); 

    for(int i = 0; i )

    // Device data allocation
    // d_ for device 
    char* d_text;
    char** d_patterns;
    int* d_pattern_lengths;
    unsigned int* d_matches_found;
    Match* d_match_arr;

    hipMalloc((void **)&d_text, text_len * sizeof(char));
    hipMalloc((void **)&d_patterns, (5+3+5)*sizeof(char));
    hipMalloc((void **)&d_pattern_lengths, 3*sizeof(int));
    hipMalloc((void **)&d_matches_found, 3*sizeof(unsigned int));
    hipMalloc((void **)&d_match_arr, 3*sizeof(Match));

    print_char_array<<<1,1>>>(d_patterns, 3);
    hipDeviceSynchronize();
    // Copy input arrays to device
    hipMemcpy(d_text, h_text, text_len * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_patterns, h_patterns, (5+3+5)*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_pattern_lengths, h_pattern_lens, 3*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_matches_found, h_matches_found, 3*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_match_arr, h_match_arr, 3*sizeof(Match), hipMemcpyHostToDevice);


    dim3 threadsPerBlock(BLOCK_SIZE);
    dim3 blocksPerGrid((ARRAY_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE);

    simple_gpu_re<<<blocksPerGrid, threadsPerBlock>>>(d_text, text_len, 3, d_patterns, d_pattern_lengths, d_matches_found, d_match_arr);

    hipMemcpy(h_match_arr, d_match_arr, 3*sizeof(Match), hipMemcpyDeviceToHost);
}