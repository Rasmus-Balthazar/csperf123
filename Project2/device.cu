#include "hip/hip_runtime.h"
#include "device.cuh"

__global__ void simple_gpu_re(char *text, int text_len, RegEx *regexes, Token *tokens, int* num_patterns, unsigned int matches_found[], Match match_arr[]) {
    // if (blockIdx.x == 0 && threadIdx.x == 0) {
    //     for (int i = 0; i < regexes[*num_patterns-1].token_offset+regexes[*num_patterns-1].token_count; i++)
    //     {
    //         printf("%c", tokens[i].to_match);
    //     }
    // }
    

    int stride = blockDim.x;
    //loop over regexes
    for (int pattern_index = blockIdx.x; pattern_index < *num_patterns; pattern_index += gridDim.x) {
        RegEx re = regexes[pattern_index];
        //find earliest pattern match
        for (int text_start = threadIdx.x; text_start < text_len; text_start += stride) {
            int token_off = 0;
            int text_off = 0;
            int does_match;
            do {
                does_match = matches(tokens+re.token_offset+token_off, text[text_start + text_off]);
                token_off += does_match;
                text_off += does_match;
                // If the token offset is longer than the amount of token we have then we have found it
                if (token_off >= re.token_count && does_match) {
                    unsigned int last_val = matches_found[pattern_index];
                    // We are relying on the checks not being exhaustive by doing val > i before atomicCAS
                    while (last_val > text_start && atomicCAS(matches_found + pattern_index, last_val, text_start) > text_start) {
                        last_val = matches_found[pattern_index];
                        // Compares b to a, and if true then writes c into a.
                    }
                    break;
                }
                if (text_start+text_off >= text_len)
                    does_match = 0;
            } while (does_match);
            
            if ((text_start + stride) > matches_found[pattern_index] || (text_start+stride) >= text_len) 
            {
                // If match here, collection process can start,
                __syncthreads(); // Synchronize threads in the block
                if (threadIdx.x == matches_found[pattern_index]%stride) {
                    match_arr[pattern_index].start_index = text_start;
                    match_arr[pattern_index].length = text_off;
                    match_arr[pattern_index].pattern_idx = pattern_index;
                    // printf("Match for pattern \"%s\" found at %i\n", patterns[pattern_index], i);
                }
                break;
            }
        }
    }
    __syncthreads();
}

// Update this to work with tokens, and return how much of text was consumed
__device__ int matches(Token *token, char text) {
    // printf("Trying to match %c and %c\n", pattern, text);
    if (token->mode)
        return 1;
    return token->to_match == text;
}


