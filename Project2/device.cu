#include "hip/hip_runtime.h"
#include "device.cuh"

__global__ void simple_gpu_re(char *text, int text_len, RegEx *regexes, Token *tokens, int* num_patterns, unsigned int matches_found[], Match match_arr[]) {
    int stride = blockDim.x;
    //loop over regexes
    for (int pattern_index = blockIdx.x; pattern_index < *num_patterns; pattern_index += gridDim.x) {
        RegEx re = regexes[pattern_index];
        //find earliest pattern match
        for (int text_start = threadIdx.x; text_start < text_len; text_start += stride) {
            int token_off = 0;
            int text_off = 0;
            int does_match;
            do {
                does_match = matches(tokens+re.token_offset+token_off, text[text_start + text_off], &token_off, text_len, text_start, &text_off);
                // If the token offset is longer than the amount of token we have then we have found it
                if (text_start+text_off >= text_len)
                    does_match = 0;
                if (token_off < 0)
                    does_match = 0;
                if (token_off >= re.token_count && does_match) {
                    unsigned int last_val = matches_found[pattern_index];
                    // We are relying on the checks not being exhaustive by doing val > i before atomicCAS
                    while (last_val > text_start && atomicCAS(matches_found + pattern_index, last_val, text_start) > text_start) {
                        last_val = matches_found[pattern_index];
                        // Compares b to a, and if true then writes c into a.
                    }
                    break;
                }
            } while (does_match);
            
            if ((text_start + stride) > matches_found[pattern_index] || (text_start+stride) >= text_len) 
            {
                // If match here, collection process can start,
                __syncthreads(); // Synchronize threads in the block
                if ((threadIdx.x == matches_found[pattern_index]%stride) && does_match) {
                    match_arr[pattern_index].start_index = text_start;
                    match_arr[pattern_index].length = text_off;
                    match_arr[pattern_index].pattern_idx = pattern_index;
                    // printf("Match for pattern \"%s\" found at %i\n", patterns[pattern_index], i);
                }
                break;
            }
        }
    }
    __syncthreads();
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        RegEx re = regexes[3];
        for (int i = 0; i < re.token_count; i++)
        {
            Token t = tokens[re.token_offset+i];
            printf("Mode: %d, Min: %d, Max: %d, to match: '%c', backtracking: %d, matched: %d", t.mode, t.min_count, t.max_count, t.to_match, t.backtracing, t.match_count);
        }
    }
}

// Update this to work with tokens, and return how much of text was consumed
__device__ int matches(Token *token, char text, int *token_off, int text_len, int text_start, int *text_off) {
    if (token->backtracing)
    {
        if (token->match_count > token->min_count) {
            token->match_count--;
            (*text_off)--;
            (*token_off)++;
            return 1;
        } else {
            token->backtracing = 0;
            (*text_off) -= token->match_count;
            (*token_off)--;
            token->match_count = 0;
            return 1;
        }
    }
    

    int text_remaining = text_len - (text_start+*text_off);
    int to_eat = min(text_remaining, token->max_count);

    // printf("Trying to match %c and %c\n", pattern, text);
    if (token->mode) {
        (*token_off) += 1;
        (*text_off) += to_eat;
        token->match_count += to_eat;
        token->backtracing = 1;
        return 1;
    } else if (token->to_match == text) {
        (*token_off)+=1;
        (*text_off)+=1;
        token->backtracing = 1;
        return 1;
    }
    (*token_off)--;
    return 1;
}


